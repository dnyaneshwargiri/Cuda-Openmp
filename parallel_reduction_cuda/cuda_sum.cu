
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#define SIZE 1024
__global__ void sum(int * A, int * C)
{
        int i=blockIdx.x*blockDim.x+threadIdx.x;
        C[i] =A[2*i+1]+A[2*i];

}

/*__global__ void avg(int * A, int * C)
{
        int i=blockIdx.x*blockDim.x+threadIdx.x;
        A[2*i] < A[2*i+1]?C[i]=A[2*i]:C[i]=A[2*i+1];

}

__global__ void stddev(int * A, int * C)
{
        int i=blockIdx.x*blockDim.x+threadIdx.x;
        A[2*i] < A[2*i+1]?C[i]=A[2*i]:C[i]=A[2*i+1];

}*/

int main()
{
        int A[SIZE];
        int *devA,*devC;
        for(int j=0;j<SIZE;j++)
        {
                A[j]=SIZE-j;
        }
        hipMalloc((void **)&devA,SIZE*sizeof(int));
        hipMalloc((void **)&devC,SIZE*sizeof(int));
        for(int j=1;j<log2((double)SIZE);j++)
        {
                hipMemcpy(devA,A,SIZE*sizeof(int),hipMemcpyHostToDevice);
                sum<<<1,SIZE/pow(2,j)>>>(devA,devC);
                hipMemcpy(&A,devC,SIZE*sizeof(int),hipMemcpyDeviceToHost);
        }
        printf("Sum is : %d",A[0]);
        hipFree(devA);
        hipFree(devC);
        return 0;

}
