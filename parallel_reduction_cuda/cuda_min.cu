
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#define SIZE 1024
__global__ void max(int * A, int * C)
{
        int i=blockIdx.x*blockDim.x+threadIdx.x;
        A[2*i] < A[2*i+1]?C[i]=A[2*i]:C[i]=A[2*i+1];

}
int main()
{
        int A[SIZE];
        int *devA,*devC;
        for(int j=0;j<SIZE;j++)
        {
                A[j]=SIZE-j;
        }
        hipMalloc((void **)&devA,SIZE*sizeof(int));
        hipMalloc((void **)&devC,SIZE*sizeof(int));
        for(int j=1;j<log2((double)SIZE);j++)
        {
                hipMemcpy(devA,A,SIZE*sizeof(int),hipMemcpyHostToDevice);
                max<<<1,SIZE/pow(2,j)>>>(devA,devC);
                hipMemcpy(&A,devC,SIZE*sizeof(int),hipMemcpyDeviceToHost);
        }
        A[0] < A[1]?printf("Min is:%d",A[0]) : printf("Min is:%d",A[1]);
        hipFree(devA);
        hipFree(devC);
        return 0;

}
